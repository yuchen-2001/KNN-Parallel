#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>

// cofig file, make changes here
#include "config.h"
#include "utils.h"

// Add CUDA error checker
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    }


/*
    ### Original version ###
    This is a 1D kernel.
    Only supports computing distances to one test point at a time.
    Only use global memory, which is very slow.

    ### Current version ###
    This kernel is a tiled GPU implementation that optimizes distance computation between all test 
    and training points using shared memory and 2D thread indexing.
    Each thread computes the distance between one training and one test point.
    Use shared memory to get high performance
*/
__global__ void batchCalcDistance (float *X_train, float *X_test, float *distance)
{
    /*
        Use shared memory to speed up repeated memory accesses
        Epecifically reduce global memory reads. Global memory much slower than shared memory.
    */

    // shared by threads in x-direction
    __shared__ float tile_train[BLOCK_X][NFEATURES];
    // shared by threads in y-direction
    __shared__ float tile_test[BLOCK_Y][NFEATURES];   

    // Fully tiled
    int train_id = blockIdx.x * blockDim.x + threadIdx.x;
    int test_id  = blockIdx.y * blockDim.y + threadIdx.y;

    // Avoiding redundant loads
    if (train_id < NTRAIN && threadIdx.y == 0) {
        for (int i = 0; i < NFEATURES; i++) {
            tile_train[threadIdx.x][i] = X_train[train_id * NFEATURES + i];
        }
    }
    if (test_id < NTEST && threadIdx.x == 0) {
        for (int i = 0; i < NFEATURES; i++) {
            tile_test[threadIdx.y][i] = X_test[test_id * NFEATURES + i];
        }
    }

    // Synchronize threads because of the use of shared memory
    __syncthreads();

    // Calculate distances
    if (train_id < NTRAIN && test_id < NTEST) {
        float dist = 0.0f;
        for (int i = 0; i < NFEATURES; ++i) {
            float diff = tile_train[threadIdx.x][i] - tile_test[threadIdx.y][i];
            dist += diff * diff;
        }
        distance[test_id * NTRAIN + train_id] = dist;
    }
}

/*
    ### Original version ###
    Sorts all NTRAIN distances for each test sample, which need O(N * log (N)) per test sample. 
    This means we need O(NTEST * N * log(N)), very slow.

    ### Current version ###
    Maintains a heap of K smallest distances
    O(N * K) per test sample, which is O(NTEST * N * K)
    Much faster when K < NTRAIN, especially for high NTRAIN. 
    (Usually the number of training data will much higher than K)

    Caution: This function only support >= 32 threads and < 1024 threads to run
    So we test 32 to 512 threads per block
*/
__global__ void findKMin(float *distances, int *min_indexes)
{
    int train_instance = blockIdx.x * blockDim.x + threadIdx.x;
    int test_instance = blockIdx.y;
    extern __shared__ float sdata[];
    int *heap_indexes = (int *)sdata;
    float *heap_distances = (float *)&heap_indexes[blockDim.x * K];
    float curr_distance;
    int curr_index;

    // Initialize each thread for size K
    if (train_instance < NTRAIN) {
        for (int i = 0; i < K; i++) {
            heap_indexes[i * blockDim.x + train_instance] = -1;
            heap_distances[i * blockDim.x + train_instance] = FLT_MAX;
        }
    }

    // wait for all threads
    __syncthreads();

    // Each thread scans its strided portion of the distance row (so threads cover the entire row in "blockDim.x" strides)
    for (int i = train_instance; i < NTRAIN; i += blockDim.x) {
        curr_distance = distances[test_instance * NTRAIN + i];
        curr_index = i;

        // Try inserting this (distance,index) into our local sorted K-array (from largest→smallest)
        for (int j = K - 1; j >= 0; j--) {
            if (heap_distances[(j * blockDim.x) + train_instance] >= curr_distance) {
                if (j == K - 1) {
                    heap_distances[(j * blockDim.x) + train_instance] = curr_distance;
                    heap_indexes[(j * blockDim.x) + train_instance] = curr_index;
                } else {
                    for (int l = K - 1; l > j; l--) {
                        heap_distances[(l * blockDim.x) + train_instance] = heap_distances[((l - 1) * blockDim.x) + train_instance];
                        heap_indexes[(l * blockDim.x) + train_instance] = heap_indexes[((l - 1) * blockDim.x) + train_instance];
                    }

                    heap_distances[(j * blockDim.x) + train_instance] = curr_distance;
                    heap_indexes[(j * blockDim.x) + train_instance] = curr_index;
                }
            }
        }
    }
    __syncthreads();

    // Every 16th thread "leader" merges K-heaps from its 16-thread subgroup
    if (threadIdx.x % 16 == 0) {
        for (int i = threadIdx.x; i < threadIdx.x + 16; i++) {
            // merge into threadIdx.x's heap
            for (int j = K - 1; j >= 0; j--) {
                if (heap_distances[(j * blockDim.x) + threadIdx.x] >= heap_distances[(j * blockDim.x) + i]) {
                    if (j == K - 1) {
                        heap_distances[(j * blockDim.x) + threadIdx.x] = heap_distances[(j * blockDim.x) + i];
                        heap_indexes[(j * blockDim.x) + threadIdx.x] = heap_indexes[(j * blockDim.x) + i];
                    } else {
                        for (int l = K - 1; l > j; l--) {
                            heap_distances[(l * blockDim.x) + threadIdx.x] = heap_distances[((l - 1) * blockDim.x) + threadIdx.x];
                            heap_indexes[(l * blockDim.x) + threadIdx.x] = heap_indexes[((l - 1) * blockDim.x) + threadIdx.x];
                        }

                        heap_distances[(j * blockDim.x) + threadIdx.x] = heap_distances[(j * blockDim.x) + i];
                        heap_indexes[(j * blockDim.x) + threadIdx.x] = heap_indexes[(j * blockDim.x) + i];
                    }
                }
            }
        }
    }
    __syncthreads();

    // Thread 0 merges the subgroup leaders into the final top-K
    if (threadIdx.x == 0) {
        for (int i = 0; i < blockDim.x / 16; i++) {
            for (int j = K - 1; j >= 0; j--) {
                if (heap_distances[(j * blockDim.x) + threadIdx.x] >= heap_distances[(j * blockDim.x) + i * 16]) {
                    if (j == K - 1) {
                        heap_distances[(j * blockDim.x) + threadIdx.x] = heap_distances[(j * blockDim.x) + i * 16];
                        heap_indexes[(j * blockDim.x) + threadIdx.x] = heap_indexes[(j * blockDim.x) + i * 16];
                    } else {
                        for (int l = K - 1; l > j; l--) {
                            heap_distances[(l * blockDim.x) + threadIdx.x] = heap_distances[((l - 1) * blockDim.x) + threadIdx.x];
                            heap_indexes[(l * blockDim.x) + threadIdx.x] = heap_indexes[((l - 1) * blockDim.x) + threadIdx.x];
                        }

                        heap_distances[(j * blockDim.x) + threadIdx.x] = heap_distances[(j * blockDim.x) + i * 16];
                        heap_indexes[(j * blockDim.x) + threadIdx.x] = heap_indexes[(j * blockDim.x) + i * 16];
                    }
                }
            }
        }

        // store final top-K indices to global memory
        for (int i = 0; i < K; i++) {
            min_indexes[test_instance * K + i] = heap_indexes[i * blockDim.x + threadIdx.x];
        }
    }
}

int predict(int *indexes, float *y_train)
{
    float* neighborCount = getFloatMat(NCLASSES, 1);
    float* probability = getFloatMat(NCLASSES, 1);

    for (int i = 0; i < NCLASSES; i++)
        neighborCount[i] = 0;

    for (int i = 0; i < K; i++) {
        int train_idx = indexes[i];
        if (train_idx < 0 || train_idx >= NTRAIN) {
            printf("Invalid index: %d (i=%d)\n", train_idx, i);
            exit(1);
        }
        int label = (int)y_train[train_idx];
        neighborCount[label]++;
    }

    for (int i = 0; i < NCLASSES; i++)
        probability[i] = neighborCount[i] / (float)K;

    int predicted_class = (int)getMax(neighborCount, NCLASSES);

    free(neighborCount);
    free(probability);

    return predicted_class;
}


int *fit(float *X_train, float *y_train, float *X_test,
    float *X_traind, float *y_traind, float *X_testd,
    float *distanced, int *min_indexes, int *min_indexesd)
{

    // Create timer event
    hipEvent_t st1, et1, st2, et2;
    float time1, time2;

    hipEventCreate(&st1);
    hipEventCreate(&et1);
    hipEventCreate(&st2);
    hipEventCreate(&et2);
    
    // Should match the whole batch of distance between test data and train data
    float *distance = getFloatMat(NTEST, NTRAIN);

    int X_train_size = sizeof(float)*NFEATURES*NTRAIN;
    int y_train_size = sizeof(float)*NTRAIN;
    int X_test_size = sizeof(float)*NFEATURES*NTEST;
    int distance_size = sizeof(float)*NTEST*NTRAIN;
    
    hipMemcpy(X_traind, X_train, X_train_size, hipMemcpyHostToDevice);
    hipMemcpy(y_traind, y_train, y_train_size, hipMemcpyHostToDevice);
    hipMemcpy(X_testd, X_test, X_test_size, hipMemcpyHostToDevice);
   
    // Number of threads in each block. 2D: BLOCK_X * BLOCK_Y
    dim3 block(BLOCK_X, BLOCK_Y);

    /*
        Number of blocks in each grid 
        Want to cover all NTRAIN * NTEST combinations using a 2D grid of blocks, where each block contains:
        BLOCK_X threads along the x-axis & BLOCK_Y threads along the y-axis
        Use (+ BLOCK_X - 1) is because we want to avoid missing data if NTRAIN isn't an exact multiple of BLOCK_X.
    */
    dim3 grid((NTRAIN + BLOCK_X - 1) / BLOCK_X, (NTEST + BLOCK_Y - 1) / BLOCK_Y);

    // Start record
    hipEventRecord(st1);

    

    /*
        Launch distance kernel 
        Use batch distance calculation
        Use 2D launch
    */
    batchCalcDistance<<<grid, block>>>(X_traind, X_testd, distanced);

    // Check CUDA
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    hipDeviceSynchronize();

    // End record
    hipEventRecord(et1);
    hipEventSynchronize(et1);
    hipEventElapsedTime(&time1, st1, et1);
    
    hipMemcpy(distance, distanced, distance_size, hipMemcpyDeviceToHost);
    hipMemcpy(distanced, distance, distance_size, hipMemcpyHostToDevice);

    /*
        Use one block per test point
        We want 1 thread block to be responsible for finding the top-K distances in that row
        This single block will collaborate via shared memory to process the entire row efficiently
    */
    dim3 gridFindKMin(1, NTEST);  


    // threads per block
    dim3 blockFindKMin(BLOCK_X * BLOCK_Y);
    
    /*
        Setup dynamic shared memory
        The kernel uses shared memory to store two arrays for each thread:
        1. distances: Each thread store K float values
        2. indexes: Each thread store K int values
        So we need (BLOCK_X * BLOCK_Y) * K * sizeof(float) + (BLOCK_X * BLOCK_Y) * K * sizeof(int)
    */
    size_t shared_mem_size = (BLOCK_X * BLOCK_Y) * K * sizeof(float) + (BLOCK_X * BLOCK_Y) * K * sizeof(int);

    
    // Start record
    hipEventRecord(st2);

    // Call sorting kernel
    findKMin<<<gridFindKMin, blockFindKMin, shared_mem_size>>>(distanced, min_indexesd);

    // Check CUDA
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // End record
    hipEventRecord(et2);
    hipEventSynchronize(et2);
    hipEventElapsedTime(&time2, st2, et2);
    
    // min_indexes stores the indices into the training set for the K nearest neighbors
    hipMemcpy(min_indexes, min_indexesd, K * NTEST * sizeof(int), hipMemcpyDeviceToHost);
    
    free(distance);

    printf("\nkernel batchCalcDistance: %.6f ms | kernel findKMin: %.6f ms\n", time1, time2);
    
    return min_indexes;
}

void readData(float **X_train, float **y_train, float **X_test, float **y_test)
{
    *X_train = initFeatures(X_TRAIN_PATH);
	*y_train = initLabels(Y_TRAIN_PATH);

	*X_test = initFeatures(X_TEST_PATH);
	*y_test = initLabels(Y_TEST_PATH);
}

int knn(float *X_train, float *y_train, float *X_test,
    float *X_traind, float *y_traind, float *X_testd,
    float *distanced, int *min_indexes, int *min_indexesd)
{

    /*
        Directly return the indexes of predictions
    */
    int *indexes = fit(X_train, y_train, X_test,
                        X_traind, y_traind, X_testd,
                        distanced, min_indexes, min_indexesd);

    int predicted_class = predict(indexes, y_train);
    free(indexes);
    return predicted_class;
}

int main()
{
    float *X_train, *y_train, *X_test, *y_test, et;
    float *X_traind, *y_traind, *X_testd, *distanced;
    int *min_indexes, *min_indexesd;

    min_indexes = (int *)calloc(NTEST * K, sizeof(int));

    // Move all memory allocation operations outside of the knn fit function
    hipMalloc((void**)&X_traind, sizeof(float)*NFEATURES*NTRAIN);
    hipMalloc((void**)&y_traind, sizeof(float)*NTRAIN);
    hipMalloc((void**)&X_testd, sizeof(float)*NFEATURES*NTEST);
    hipMalloc((void**)&distanced, sizeof(float)*NTRAIN*NTEST);
    hipMalloc((void**)&min_indexesd, sizeof(int) * NTEST * K);
    
    hipEvent_t start, stop;
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipEventRecord(start);
 
    
    //read data
    readData(&X_train, &y_train, &X_test, &y_test);
    
    //call knn
    int predicted_class = knn(X_train, y_train, X_test,
        X_traind, y_traind, X_testd,
        distanced, min_indexes, min_indexesd);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&et, start, stop);
   
    printf("Time taken: %fms\n", et);
    
    
    // printf("Predicted label: %d True label: %d\n", predicted_class, (int)y_test[randId]);
    // Free the allocated memory
    hipFree(X_traind);
    hipFree(y_traind);
    hipFree(X_testd);
    hipFree(distanced);
    hipFree(min_indexesd);
     
	free(X_train);
	free(y_train);

	free(X_test);
	free(y_test);
    
    return 0;
}
